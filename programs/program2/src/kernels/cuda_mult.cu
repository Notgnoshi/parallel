#include "hip/hip_runtime.h"
#include "kernels/cuda_mult.h"
#include "matrix.h"
#include "validator.h"
#include <iostream>

//! @brief Use @f$16 \times 16@f$ blocks of threads.
//! @details Use a `#define` so that it's accessable in both device and host code.
#define BLOCK_XDIM 16
#define BLOCK_YDIM 16
#define BLOCK_ZDIM 1
//! @todo Document.
static const dim3 BLOCK_SIZE( BLOCK_XDIM, BLOCK_YDIM, BLOCK_ZDIM );

/**
 * @brief A trivially easy-to-copy struct to hold the matrices on the device.
 *
 * @details The Matrix_t struct defines a copy-constructor, destructor, etc, so
 * it doesn't work well for passing to CUDA kernel calls. So use this struct to
 * hold trivially-easy-to-copy device fields so that the kernel calls don't get
 * so nasty.
 */
struct DeviceMatrix_t
{
    //! @brief The width of the matrix.
    size_t cols;
    //! @brief The height of the matrix.
    size_t rows;
    //! @brief The stride in the 1D array that the 2D data is stored in.
    size_t stride;
    //! @brief The matrix data stored in row-major order.
    double* data;
};

/**
 * @brief Given a matrix, get the element at (row, col).
 *
 * @param matrix The DeviceMatrix_t holding the data to access.
 * @param row ...
 * @param col ...
 * @returns the element at the given location.
 */
__device__ static double GetElement( const DeviceMatrix_t matrix, size_t row, size_t col )
{
    return matrix.data[row * matrix.stride + col];
}

/**
 * @brief Given a matrix, set the value at (row, col).
 *
 * @param[out] matrix The DeviceMatrix_t matrix to modify.
 * @param row ...
 * @param col ...
 * @param value The value to insert at (row, col)
 */
__device__ static void SetElement( DeviceMatrix_t matrix, size_t row, size_t col, double value )
{
    matrix.data[row * matrix.stride + col] = value;
}

/**
 * @brief Get the @f$(row, col)@f$th sub matrix in the given matrix.
 *
 * @param matrix The matrix to extract sub matrices from.
 * @param row Which row of sub-matrices to extract the sub matrix from.
 * @param col Which column of sub-matrices to extract the sub matrix from.
 * @returns A submatrix from the given matrix.
 */
__device__ static DeviceMatrix_t GetSubMatrix( const DeviceMatrix_t matrix, size_t row, size_t col )
{
    DeviceMatrix_t sub;
    //! @todo for non-square blocks, these will flip depending on lhs or rhs.
    sub.cols = BLOCK_XDIM;
    sub.rows = BLOCK_YDIM;
    sub.stride = matrix.stride;
    //! @todo Figure out if it's (x, y) or (y, x) for non-square blocks.
    sub.data = &matrix.data[matrix.stride * BLOCK_YDIM * row + BLOCK_XDIM * col];

    return sub;
}

/**
 * @brief The multiplication kernel.
 *
 * @note Due to CUDA limitations, this cannot be a private method in the CudaAdditionKernel
 * class. Further, note that it is not possible to pass by reference to a CUDA kernel.
 * Meaning that the structure copy constructor is called every time the kernel is
 * called, and the destructor is called every time each thread finishes running
 * the kernel. This has disasterous consequences if the destructor attempts to
 * do RAII and deletes the data pointer. So the solution is to not pass structs
 * to the kernel, but instead their data pointers.
 *
 * @details Using the `MatMulKernel` in the CUDA documentation for inspiration,
 * this kernel uses shared memory and submatrices to implement matrix multiplication.
 * The biggest difference between this implementation and that in the CUDA
 * documentation is that this will work for matrices that are not evenly divisible
 * by the block size.
 *
 * @see https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#shared-memory__matrix-multiplication-shared-memory for a good picture of what I'm doing.
 *
 * @todo Document.
 */
__global__ static void MultiplicationKernel( const DeviceMatrix_t lhs, const DeviceMatrix_t rhs, DeviceMatrix_t result )
{
    // Each block of threads computes a submatrix of the result.
    DeviceMatrix_t sub = GetSubMatrix( result, blockIdx.y, blockIdx.x );

    // Each thread computes one value in the submatrix.
    double value = 0;

    // Loop over the submatrices of lhs and rhs to compute sub.
    for( size_t i = 0; i < ( BLOCK_XDIM + lhs.cols - 1 ) / BLOCK_XDIM; ++i )
    {
        // Fixed row and variable columm.
        const DeviceMatrix_t lhs_sub = GetSubMatrix( lhs, blockIdx.y, i );
        // Fixed column and variable row.
        const DeviceMatrix_t rhs_sub = GetSubMatrix( rhs, i, blockIdx.x );

        // Share lhs_sub and rhs_sub across the block.
        __shared__ double left_block[BLOCK_YDIM][BLOCK_XDIM];
        __shared__ double right_block[BLOCK_YDIM][BLOCK_XDIM];

        //! @note Each thread loads an element of the result submatrix that their
        //! block is responsible for computing. However, note that since the blocks
        //! can cover more than the matrix memory, this will access memory we do
        //! not own. This is bad, but I'd rather do that than add more branches.
        //! We don't modify the data, so it's not *that* bad... (it actually is).
        left_block[threadIdx.y][threadIdx.x] = GetElement( lhs_sub, threadIdx.y, threadIdx.x );
        right_block[threadIdx.y][threadIdx.x] = GetElement( rhs_sub, threadIdx.y, threadIdx.x );

        // Finish loading the shared memory before proceeding.
        __syncthreads();

        //! @note You have no idea how long it took, or how many pages of graph paper
        //! I went through to get this formula.
        const size_t lim = BLOCK_YDIM - ( ( ( i + 1 ) * BLOCK_YDIM ) % lhs.cols ) % BLOCK_YDIM;
        for( size_t j = 0; j < lim; ++j )
        {
            value += left_block[threadIdx.y][j] * right_block[j][threadIdx.x];
        }

        // Make sure computation is finished before loading new submatrices into shared memory.
        __syncthreads();
    }

    // Convert coordinates of the submatrix and the location inside the submatrix
    // into the coordinates of the whole result matrix.
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;

    // Only set the submatrix value if it lies inside the result.
    if( row < result.rows && col < result.cols )
    {
        // Each thread writes one element in their block's submatrix.
        SetElement( sub, threadIdx.y, threadIdx.x, value );
    }
}

std::shared_ptr<Matrix_t> CudaMultiplicationKernel::Operation( const Matrix_t& lhs, const Matrix_t& rhs )
{
    if( !MultiplicationValidator( lhs, rhs ) )
    {
        // std::cerr << "Dimensions (" << lhs.rows << ", " << lhs.cols << ")"
        //           << " * (" << rhs.rows << ", " << rhs.cols << ")"
        //           << " incompatible for multiplication." << std::endl;
        return std::make_shared<Matrix_t>( 0, 0 );
    }

    // Allocate memory for the result on the host.
    auto result = std::make_shared<Matrix_t>( lhs.rows, rhs.cols );

    DeviceMatrix_t _lhs;
    DeviceMatrix_t _rhs;
    DeviceMatrix_t _result;

    _lhs.cols = lhs.cols;
    _lhs.stride = lhs.cols;
    _lhs.rows = lhs.rows;
    _rhs.cols = rhs.cols;
    _rhs.stride = rhs.cols;
    _rhs.rows = rhs.rows;
    _result.cols = result->cols;
    _result.stride = result->cols;
    _result.rows = result->rows;

    hipMalloc( &_result.data, result->elements * sizeof( double ) );
    hipMalloc( &_lhs.data, lhs.elements * sizeof( double ) );
    hipMalloc( &_rhs.data, rhs.elements * sizeof( double ) );

    // Copy the operands to the device.
    hipMemcpy( _lhs.data, lhs.data, lhs.elements * sizeof( double ), hipMemcpyHostToDevice );
    hipMemcpy( _rhs.data, rhs.data, rhs.elements * sizeof( double ), hipMemcpyHostToDevice );

    dim3 grid_size(
        ( BLOCK_SIZE.x + result->cols - 1 ) / BLOCK_SIZE.x,
        ( BLOCK_SIZE.y + result->rows - 1 ) / BLOCK_SIZE.y,
        1 );
    MultiplicationKernel<<<grid_size, BLOCK_SIZE>>>( _lhs, _rhs, _result );

    hipDeviceSynchronize();

    // Copy the result from the device to the host.
    hipMemcpy( result->data, _result.data, result->elements * sizeof( double ), hipMemcpyDeviceToHost );

    // Every good programmer knows every malloc() should have a corresponding free().
    hipFree( _result.data );
    hipFree( _lhs.data );
    hipFree( _rhs.data );

    return result;
}
