#include "hip/hip_runtime.h"
#include "kernels/cuda_mult.h"
#include "matrix.h"
#include "validator.h"
#include <iostream>

//! @brief Use @f$16 \times 16@f$ blocks of threads.
//! @details Use a `#define` so that it's accessable in both device and host code.
#define BLOCK_XDIM 16
#define BLOCK_YDIM 16
#define BLOCK_ZDIM 1
//! @todo Document.
static const dim3 BLOCK_SIZE( BLOCK_XDIM, BLOCK_YDIM, BLOCK_ZDIM );

/**
 * @brief A trivially easy-to-copy struct to hold the matrices on the device.
 *
 * @details The Matrix_t struct defines a copy-constructor, destructor, etc, so
 * it doesn't work well for passing to CUDA kernel calls. So use this struct to
 * hold trivially-easy-to-copy device fields so that the kernel calls don't get
 * so nasty.
 */
struct DeviceMatrix_t
{
    //! @brief The width of the matrix.
    size_t width;
    //! @brief The height of the matrix.
    size_t height;
    //! @brief The tile stride for sub matrices of the matrix.
    size_t stride;
    //! @brief The matrix data stored in row-major order.
    double* data;
};

/**
 * @brief Given a matrix, get the element at (row, col).
 *
 * @param matrix The DeviceMatrix_t holding the data to access.
 * @param row ...
 * @param col ...
 * @returns the element at the given location.
 */
__device__ static double GetElement( const DeviceMatrix_t matrix, size_t row, size_t col )
{
    return matrix.data[row * matrix.stride + col];
}

/**
 * @brief Given a matrix, set the value at (row, col).
 *
 * @param[out] matrix The DeviceMatrix_t matrix to modify.
 * @param row ...
 * @param col ...
 * @param value The value to insert at (row, col)
 */
__device__ static void SetElement( DeviceMatrix_t matrix, size_t row, size_t col, double value )
{
    matrix.data[row * matrix.stride + col] = value;
}

/**
 * @brief Get the @f$(row, col)@f$th sub matrix in the given matrix.
 *
 * @param matrix The matrix to extract sub matrices from.
 * @param row Which row of sub-matrices to extract the sub matrix from.
 * @param col Which column of sub-matrices to extract the sub matrix from.
 * @returns A submatrix from the given matrix.
 */
__device__ static DeviceMatrix_t GetSubMatrix( DeviceMatrix_t matrix, size_t row, size_t col )
{
    DeviceMatrix_t sub;
    //! @todo for non-square blocks, these will flip depending on lhs or rhs.
    sub.width = BLOCK_XDIM;
    sub.height = BLOCK_YDIM;
    sub.stride = matrix.stride;
    //! @todo Figure out if it's (x, y) or (y, x) for non-square blocks.
    sub.data = &matrix.data[matrix.stride * BLOCK_XDIM * row + BLOCK_YDIM * col];

    return sub;
}

/**
 * @brief The multiplication kernel.
 *
 * @note Due to CUDA limitations, this cannot be a private method in the CudaAdditionKernel
 * class. Further, note that it is not possible to pass by reference to a CUDA kernel.
 * Meaning that the structure copy constructor is called every time the kernel is
 * called, and the destructor is called every time each thread finishes running
 * the kernel. This has disasterous consequences if the destructor attempts to
 * do RAII and deletes the data pointer. So the solution is to not pass structs
 * to the kernel, but instead their data pointers.
 *
 * @see https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#shared-memory__matrix-multiplication-shared-memory for details.
 *
 * @todo Document.
 */
__global__ static void MultiplicationKernel( DeviceMatrix_t lhs, DeviceMatrix_t rhs, DeviceMatrix_t result )
{
    // Each block of threads computes a submatrix of the result.
    DeviceMatrix_t sub = GetSubMatrix( result, blockIdx.y, blockIdx.x );
    // Each thread computes one value in the submatrix.
    double value = 0;

    //! @todo Only set the submatrix value if it lies inside the result.

    // Loop over the submatrices of lhs and rhs to compute sub.
    for( size_t i = 0; i < ( BLOCK_XDIM + lhs.width - 1 ) / BLOCK_XDIM; ++i )
    {
        DeviceMatrix_t lhs_sub = GetSubMatrix( lhs, blockIdx.y, i );
        DeviceMatrix_t rhs_sub = GetSubMatrix( rhs, i, blockIdx.x );

        // Share lhs_sub and rhs_sub across the block.
        __shared__ double left_block[BLOCK_XDIM][BLOCK_YDIM];
        __shared__ double right_block[BLOCK_XDIM][BLOCK_YDIM];

        //! @todo Only set the values of the submatrices if they exist (the blocks can
        // overlap the edges of the matrices).

        // Each thread will load a single element of each submatrix into the shared memory.
        left_block[threadIdx.y][threadIdx.x] = GetElement(lhs_sub, threadIdx.y, threadIdx.x);
        right_block[threadIdx.y][threadIdx.x] = GetElement(rhs_sub, threadIdx.y, threadIdx.x);

        // Finish loading the shared memory before proceeding.
        __syncthreads();

        //! @todo This assumes square blocks?
        for( size_t j = 0; j < BLOCK_YDIM; ++j )
        {
            value += left_block[threadIdx.y][j] * right_block[j][threadIdx.x];
        }

        // Make sure computation is finished before loading new submatrices into shared memory.
        __syncthreads();
    }

    // Each thread writes one element in their block's submatrix.
    SetElement( sub, threadIdx.y, threadIdx.x, value );
}

std::shared_ptr<Matrix_t> CudaMultiplicationKernel::Operation( const Matrix_t& lhs, const Matrix_t& rhs )
{
    if( !MultiplicationValidator( lhs, rhs ) )
    {
        // std::cerr << "Dimensions (" << lhs.rows << ", " << lhs.cols << ")"
        //           << " * (" << rhs.rows << ", " << rhs.cols << ")"
        //           << " incompatible for multiplication." << std::endl;
        return std::make_shared<Matrix_t>( 0, 0 );
    }

    // Allocate memory for the result on the host.
    auto result = std::make_shared<Matrix_t>( lhs.rows, rhs.cols );

    DeviceMatrix_t _lhs;
    DeviceMatrix_t _rhs;
    DeviceMatrix_t _result;

    _lhs.width = lhs.cols;
    _lhs.stride = lhs.cols;
    _lhs.height = lhs.rows;
    _rhs.width = rhs.cols;
    _rhs.stride = rhs.cols;
    _rhs.height = rhs.rows;
    _result.width = result->cols;
    _result.stride = result->cols;
    _result.height = result->rows;

    hipMalloc( &_result.data, result->elements * sizeof( double ) );
    hipMalloc( &_lhs.data, lhs.elements * sizeof( double ) );
    hipMalloc( &_rhs.data, rhs.elements * sizeof( double ) );

    // Copy the operands to the device.
    hipMemcpy( _lhs.data, lhs.data, lhs.elements * sizeof( double ), hipMemcpyHostToDevice );
    hipMemcpy( _rhs.data, rhs.data, rhs.elements * sizeof( double ), hipMemcpyHostToDevice );

    //! @todo Handle matrices not evenly divisible by the block size.
    dim3 grid_size( rhs.cols / BLOCK_SIZE.x, lhs.rows / BLOCK_SIZE.y, 1 );
    MultiplicationKernel<<<grid_size, BLOCK_SIZE>>>( _lhs, _rhs, _result );

    hipDeviceSynchronize();

    // Copy the result from the device to the host.
    hipMemcpy( result->data, _result.data, result->elements * sizeof( double ), hipMemcpyDeviceToHost );

    // Every good programmer knows every malloc() should have a corresponding free().
    hipFree( _result.data );
    hipFree( _lhs.data );
    hipFree( _rhs.data );

    return result;
}
