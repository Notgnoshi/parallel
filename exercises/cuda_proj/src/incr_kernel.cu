#include "hip/hip_runtime.h"
#include "incr_kernel.cuh"
#include "incr_wrapper.h"

__global__ void incr_kernel( double* a, const double* b, size_t n )
{
    for( size_t i = 0; i < n; ++i )
    {
        a[i] = a[i] + b[i];
    }
}

void incr( double* a, const double* b, size_t n )
{
    double* _a;
    double* _b;

    hipMalloc( &_a, n * sizeof( double ) );
    hipMalloc( &_b, n * sizeof( double ) );

    hipMemcpy( _a, a, n * sizeof( double ), hipMemcpyHostToDevice );
    hipMemcpy( _b, b, n * sizeof( double ), hipMemcpyHostToDevice );

    //! @note This is essentially serial, with *loads* of memcpy overhead.
    incr_kernel<<<1, 1>>>( _a, _b, n );

    hipMemcpy( a, _a, n * sizeof( double ), hipMemcpyDeviceToHost );

    hipFree( _a );
    hipFree( _b );
}
